/**
 * @file interposer.c
 * @brief Demonstrates using of libcudainterpose.c
 *
 * @date Feb 4, 2011
 * @author Magda S., magg@gatech.edu
 */

//#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
		a[idx] = a[idx] * a[idx];
}


// main routine that executes on the host
// >>>>>>>>> change main() -> cuda_main()
void* cuda_main(void* t) {
	long int gpuid = (long int)t;
	float *a_h, *a_d; // Pointer to host & device arrays
	const int N = 10; // Number of elements in arrays
	size_t size = N * sizeof(float);
	int i;
	int deviceCount = 0;
	hipDeviceProp_t prop;
	int cur_dev;
        

	hipGetDeviceCount(&deviceCount);
        printf("NO OF DEVICES: %d\n", deviceCount);

	if(hipSetDevice(gpuid)!=hipSuccess) printf("FAILED\n");
	hipGetDevice(&cur_dev);
	printf("Dev: %d\n", cur_dev);


/*	cudaGetDeviceCount(&deviceCount);

	for(i = 0; i < deviceCount; i++){
		cudaGetDeviceProperties(&prop, i);
	}
*/
//	printf("%s.%d: The number of cuda devices is %d\n", __FUNCTION__, __LINE__, deviceCount);

	a_h = (float *) malloc(size); // Allocate array on host
	a_d = NULL;

	hipMalloc((void **) &a_d, size); // Allocate array on device
	printf("after cudaMalloc: a_d = %p\n", a_d);
	// Initialize host array and copy it to CUDA device
	for (int i = 0; i < N; i++)
		a_h[i] = (float) i;
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	// Do calculation on device:
	int block_size = 4;
	int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);

	square_array <<< n_blocks, block_size >>> (a_d, N);
	//cudaConfigureCall(n_blocks, block_size, 0, 0);
	//cudaSetupArgument(&a_d, sizeof(float*), 0);
	//cudaSetupArgument(&N, sizeof(int), 0);
	//cudaLaunch("square_array");	

	// Retrieve result from device and store it in host array
	hipMemcpy(a_h, a_d, sizeof(float) * N, hipMemcpyDeviceToHost);
	// Print results
	for (int i = 0; i < N; i++)
		printf("%d %f\n", i, a_h[i]);

	// Cleanup
	free(a_h);
	hipFree(a_d);

//------------------------------------------------
#if 0
        cudaSetDevice(2);
        cudaGetDevice(&cur_dev);
        printf("Dev: %d\n", cur_dev);


/*      cudaGetDeviceCount(&deviceCount);

        for(i = 0; i < deviceCount; i++){
                cudaGetDeviceProperties(&prop, i);
        }
*/
//      printf("%s.%d: The number of cuda devices is %d\n", __FUNCTION__, __LINE__, deviceCount);

        a_h = (float *) malloc(size); // Allocate array on host
        a_d = NULL;

        cudaMalloc((void **) &a_d, size); // Allocate array on device
        printf("after cudaMalloc: a_d = %p\n", a_d);
        // Initialize host array and copy it to CUDA device
        for (int i = 0; i < N; i++)
                a_h[i] = (float) i;
        cudaMemcpy(a_d, a_h, size, cudaMemcpyHostToDevice);
        // Do calculation on device:
        block_size = 4;
        n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
        square_array <<< n_blocks, block_size >>> (a_d, N);
        // Retrieve result from device and store it in host array
        cudaMemcpy(a_h, a_d, sizeof(float) * N, cudaMemcpyDeviceToHost);
        // Print results
        for (int i = 0; i < N; i++)
                printf("%d %f\n", i, a_h[i]);

        // Cleanup
        free(a_h);
        cudaFree(a_d);
#endif





	return 0;
}

int main(){
	// I wonder, if this cannot be done by Python, since this is administration
	// and python integrates with c, so I think it might be wiser
	// to use python for that. but maybe later, when the thinks will clarify
	// create GPU assembly
	// 0. init if not initialized
	// (not here outthere) the device - create in buStore the representations
	// of the physical devices
	// 1. specify how many GPU you need
	// 2. create as many vgpu (in terms of structures) as required
	// 3. wire vgpu to gpus
	// 4. gpu assembly
	// 5. some process needs to clean up after - but we do not worry about that

	printf("\n*******************************\n");
	printf("CUDA APP START\n");
	printf("*******************************\n");
	
	pthread_t thid1, thid2;
	// now call the cuda main, so our GA enabler library can sort out
	// which cuda call goes where
	pthread_create(&thid1, NULL, cuda_main, (void*)0);
	pthread_create(&thid2, NULL, cuda_main, (void*)2);
	//cuda_main();
	pthread_join(thid1, NULL);
	pthread_join(thid2, NULL);


	return 0;
}
