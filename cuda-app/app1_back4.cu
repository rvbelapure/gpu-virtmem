#include "hip/hip_runtime.h"
/**
 * @file interposer.c
 * @brief Demonstrates using of libcudainterpose.c
 *
 * @date Feb 4, 2011
 * @author Magda S., magg@gatech.edu
 */

//#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <assert.h>

#define ARRAY_SIZE 1024
int els_for_each;
float* a_h_global;

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
		a[idx] = a[idx] * a[idx];
}


// main routine that executes on the host
// >>>>>>>>> change main() -> cuda_main()
void* cuda_main(void* t) {
	long int gpuid = (long int)t;
	float *a_h, *a_d; // Pointer to host & device arrays
	const int N = els_for_each; // Number of elements in arrays
	size_t size = N * sizeof(float);
	int i;
	int deviceCount = 0;
	hipDeviceProp_t prop;
	int cur_dev;
        


	assert(hipSetDevice(gpuid)==hipSuccess) ;
	hipGetDevice(&cur_dev);
	printf("Dev: %d\n", cur_dev);


/*	hipGetDeviceCount(&deviceCount);

	for(i = 0; i < deviceCount; i++){
		hipGetDeviceProperties(&prop, i);
	}
*/
//	printf("%s.%d: The number of cuda devices is %d\n", __FUNCTION__, __LINE__, deviceCount);

	a_h = a_h_global + gpuid*els_for_each; // Allocate array on host
	a_d = NULL;

	assert(hipMalloc((void **) &a_d, size)==hipSuccess); // Allocate array on device
	printf("after hipMalloc: a_d = %p\n", a_d);
	// Initialize host array and copy it to CUDA device
	//for (int i = 0; i < N; i++)
	//	a_h[i] = (float) i;
	assert(hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice)==hipSuccess);
	// Do calculation on device:
	int block_size = 16;
	int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);

	assert(square_array <<< n_blocks, block_size >>> (a_d, N) == hipSuccess);
	//hipConfigureCall(n_blocks, block_size, 0, 0);
	//hipSetupArgument(&a_d, sizeof(float*), 0);
	//hipSetupArgument(&N, sizeof(int), 0);
	//hipLaunchByPtr("square_array");	

	// Retrieve result from device and store it in host array
	assert(hipMemcpy(a_h, a_d, sizeof(float) * N, hipMemcpyDeviceToHost) == hipSuccess);
	
	// Cleanup
	//free(a_h);
	hipFree(a_d);

//------------------------------------------------
#if 0
        hipSetDevice(2);
        hipGetDevice(&cur_dev);
        printf("Dev: %d\n", cur_dev);


/*      hipGetDeviceCount(&deviceCount);

        for(i = 0; i < deviceCount; i++){
                hipGetDeviceProperties(&prop, i);
        }
*/
//      printf("%s.%d: The number of cuda devices is %d\n", __FUNCTION__, __LINE__, deviceCount);

        a_h = (float *) malloc(size); // Allocate array on host
        a_d = NULL;

        hipMalloc((void **) &a_d, size); // Allocate array on device
        printf("after hipMalloc: a_d = %p\n", a_d);
        // Initialize host array and copy it to CUDA device
        for (int i = 0; i < N; i++)
                a_h[i] = (float) i;
        hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
        // Do calculation on device:
        block_size = 4;
        n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
        square_array <<< n_blocks, block_size >>> (a_d, N);
        // Retrieve result from device and store it in host array
        hipMemcpy(a_h, a_d, sizeof(float) * N, hipMemcpyDeviceToHost);
        // Print results
        for (int i = 0; i < N; i++)
                printf("%d %f\n", i, a_h[i]);

        // Cleanup
        free(a_h);
        hipFree(a_d);
#endif





	return 0;
}

int main(){
	// I wonder, if this cannot be done by Python, since this is administration
	// and python integrates with c, so I think it might be wiser
	// to use python for that. but maybe later, when the thinks will clarify
	// create GPU assembly
	// 0. init if not initialized
	// (not here outthere) the device - create in buStore the representations
	// of the physical devices
	// 1. specify how many GPU you need
	// 2. create as many vgpu (in terms of structures) as required
	// 3. wire vgpu to gpus
	// 4. gpu assembly
	// 5. some process needs to clean up after - but we do not worry about that

	printf("\n*******************************\n");
	printf("CUDA APP START\n");
	printf("*******************************\n");
	
	a_h_global;
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
        printf("NO OF DEVICES: %d\n", deviceCount);

	a_h_global = (float *) malloc(ARRAY_SIZE*sizeof(float)); // Allocate array on host
	for (int i = 0; i <ARRAY_SIZE ; i++)
                a_h_global[i] = (float) i;
	pthread_t* thid_array=(pthread_t*)malloc(deviceCount*sizeof(pthread_t));
	
	
	// now call the cuda main, so our GA enabler library can sort out
	// which cuda call goes where
	
	els_for_each = ARRAY_SIZE/deviceCount;

	for(int i =0; i<deviceCount; i++)
	   pthread_create(&thid_array[i], NULL, cuda_main, (void*)i);
	
	//cuda_main();
	for(int i =0; i<deviceCount; i++)
	   pthread_join(thid_array[i], NULL);
	// Print results
	for (int i = 0; i < ARRAY_SIZE; i++)
		printf("%d %f\n", i, a_h_global[i]);



	return 0;
}
