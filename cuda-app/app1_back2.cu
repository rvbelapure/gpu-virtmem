/**
 * @file interposer.c
 * @brief Demonstrates using of libcudainterpose.c
 *
 * @date Feb 4, 2011
 * @author Magda S., magg@gatech.edu
 */

//#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
		a[idx] = a[idx] * a[idx];
}

// main routine that executes on the host
// >>>>>>>>> change main() -> cuda_main()
int cuda_main(void) {
	float *a_h, *a_h2, *a_d, *a_d2; // Pointer to host & device arrays
	const int N = 10; // Number of elements in arrays
	size_t size = N * sizeof(float);
	int i;
	int deviceCount = 0;
	hipDeviceProp_t prop;
	int cur_dev;

	if(hipSetDevice(1) != hipSuccess) printf("FAILED\n");
	hipGetDevice(&cur_dev);
	printf("Dev: %d\n", cur_dev);


/*	cudaGetDeviceCount(&deviceCount);

	for(i = 0; i < deviceCount; i++){
		cudaGetDeviceProperties(&prop, i);
	}
*/
//	printf("%s.%d: The number of cuda devices is %d\n", __FUNCTION__, __LINE__, deviceCount);

	a_h = (float *) malloc(size); // Allocate array on host
	a_d = NULL;

	hipMalloc((void **) &a_d, size); // Allocate array on device
	printf("after cudaMalloc: a_d = %p\n", a_d);
	// Initialize host array and copy it to CUDA device
	for (int i = 0; i < N; i++)
		a_h[i] = (float) i;
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	// Do calculation on device:
	int block_size = 4;
	int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);

	square_array <<< n_blocks, block_size >>> (a_d, N);
	//cudaConfigureCall(n_blocks, block_size, 0, 0);
	//cudaSetupArgument(&a_d, sizeof(float*), 0);
	//cudaSetupArgument(&N, sizeof(int), 0);
	//cudaLaunch("square_array");	

//------------------------------------------------

        if(hipSetDevice(0) != hipSuccess) printf("FAILED\n");;
        hipGetDevice(&cur_dev);
        printf("Dev: %d\n", cur_dev);


/*      cudaGetDeviceCount(&deviceCount);

        for(i = 0; i < deviceCount; i++){
                cudaGetDeviceProperties(&prop, i);
        }
*/
//      printf("%s.%d: The number of cuda devices is %d\n", __FUNCTION__, __LINE__, deviceCount);

        a_h2 = (float *) malloc(size); // Allocate array on host
        a_d2 = NULL;

        hipMalloc((void **) &a_d2, size); // Allocate array on device
        printf("after cudaMalloc: a_d = %p\n", a_d);
        // Initialize host array and copy it to CUDA device
        for (int i = 0; i < N; i++)
                a_h2[i] = (float) i+1;
        hipMemcpy(a_d2, a_h2, size, hipMemcpyHostToDevice);
        // Do calculation on device:
        block_size = 4;
        n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
        square_array <<< n_blocks, block_size >>> (a_d2, N);

	hipSetDevice(1);
	// Retrieve result from device and store it in host array
	hipMemcpy(a_h, a_d, sizeof(float) * N, hipMemcpyDeviceToHost);
	// Print results
	for (int i = 0; i < N; i++)
		printf("%d %f\n", i, a_h[i]);

	// Cleanup
	free(a_h);
	hipFree(a_d);

	hipSetDevice(0);
        // Retrieve result from device and store it in host array
        hipMemcpy(a_h2, a_d2, sizeof(float) * N, hipMemcpyDeviceToHost);
        // Print results
        for (int i = 0; i < N; i++)
                printf("%d %f\n", i, a_h2[i]);

        // Cleanup
        free(a_h);
        hipFree(a_d);






	return 0;
}

int main(){
	// I wonder, if this cannot be done by Python, since this is administration
	// and python integrates with c, so I think it might be wiser
	// to use python for that. but maybe later, when the thinks will clarify
	// create GPU assembly
	// 0. init if not initialized
	// (not here outthere) the device - create in buStore the representations
	// of the physical devices
	// 1. specify how many GPU you need
	// 2. create as many vgpu (in terms of structures) as required
	// 3. wire vgpu to gpus
	// 4. gpu assembly
	// 5. some process needs to clean up after - but we do not worry about that

	printf("\n*******************************\n");
	printf("CUDA APP START\n");
	printf("*******************************\n");

	// now call the cuda main, so our GA enabler library can sort out
	// which cuda call goes where
	cuda_main();

	return 0;
}
