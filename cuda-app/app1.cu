#include "hip/hip_runtime.h"
/**
 * @file interposer.c
 * @brief Demonstrates using of libcudainterpose.c
 *
 * @date Feb 4, 2011
 * @author Magda S., magg@gatech.edu
 */

//#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include "particle.h"
#include <assert.h>

#define SWIDTH 400
#define SHEIGHT 400
#define DT .5

int els_for_each;
float* a_h_global;
Particle *parts;
int nIters, pNum;
int deviceCount;
pthread_barrier_t barr;



__global__
void checkCollisionWithParticles(Particle *parts, int n, float dt, int gpuid, int BLOCK_SIZE, int els_for_each)
{

        int thid=blockIdx.x*BLOCK_SIZE+threadIdx.x;


	int i=thid+els_for_each*gpuid;
	{
		Vec3 p1,p2;
		Vec3 v1,v2;
		float m1;
		
		m1 = parts[i].m;
		
		p1[0] = parts[i].p[0]+parts[i].v[0]*dt;
		p1[1] = parts[i].p[1]+parts[i].v[1]*dt;
		p1[2] = parts[i].p[2]+parts[i].v[2]*dt;
		
		v1[0] = parts[i].v[0];
		v1[1] = parts[i].v[1];
		v1[2] = parts[i].v[2];
		
		for(int j=0; j<n;j++)
		{
			if(i!=j && !parts[j].willCollide && !parts[i].willCollide)
			{
				p2[0] = parts[j].p[0]+parts[j].v[0]*dt;
				p2[1] = parts[j].p[1]+parts[j].v[1]*dt;
				p2[2] = parts[j].p[2]+parts[j].v[2]*dt;
				
				v2[0] = parts[j].v[0];
				v2[1] = parts[j].v[1];
				v2[2] = parts[j].v[2];
				if(dist(p2,p1)<parts[i].r+parts[j].r)
				{
					Vec3 auxx, auxy, auxz;
					float v1xp, v2xp, v1x, v1y, v2x, v2y, m2 = parts[j].m, A, B, a, b, c;
					for(int k=0; k<3; k++)
						auxx[k] = parts[j].p[k]-parts[i].p[k];
						
					norm(auxx,&auxx);
					auxy[0] = -auxx[1];
					auxy[1] = auxx[0];
					
					auxz[0] = auxz[1] = 0;
					auxz[2] = 1;
					
					v1x = dot(parts[i].v,auxx);
					v1y = dot(parts[i].v,auxy);
					v2x = dot(parts[j].v,auxx);
					v2y = dot(parts[j].v,auxy);
					
					//A = v1x + (m2/m1)*v2x;
					//B = v1x*v1x + (m2/m1)*v2x*v2x;
					
					//a = m2/m1 + (m2*m2)/(m1*m1);
					//b = -2*A*(m2/m1);
					//c = A*A-B;
					
					//v2xp = (-b+sqrt(b*b-4*a*c))/(2*a);
					//v1xp = A-(m2/m1)*v2xp;
					v1xp = ((m1-m2)*v1x+2*m2*v2x)/(m1+m2);
					v2xp = v1x-v2x+v1xp;
					
					parts[i].willCollide = true;
					parts[j].willCollide = true;
									
					parts[i].nv[0] = auxx[0]*v1xp + auxy[0]*v1y;
					parts[i].nv[1] = auxx[1]*v1xp + auxy[1]*v1y;
					parts[i].nv[2] = auxx[2]*v1xp + auxy[2]*v1y;
					
					parts[j].nv[0] = auxx[0]*v2xp + auxy[0]*v2y;
					parts[j].nv[1] = auxx[1]*v2xp + auxy[1]*v2y;
					parts[j].nv[2] = auxx[2]*v2xp + auxy[2]*v2y;
					
				}
			}
		}
	}
}


__global__
void checkCollisionWithBox(Particle *parts, int n, float dt, int gpuid, int BLOCK_SIZE, int els_for_each)
{

        int thid=blockIdx.x*BLOCK_SIZE+threadIdx.x;


	int i=thid+els_for_each*gpuid;
	{
            if(!parts[i].willCollide)
            {
		float colt;
		float colv;
		if(parts[i].p[0]+parts[i].r+parts[i].v[0]*dt>SWIDTH/2.0 || parts[i].p[0]-parts[i].r+parts[i].v[0]*dt<-SWIDTH/2.0)
		{
			//colt = (SWIDTH/2.0-fabs(parts[i].p[0])-parts[i].r)/parts[i].v[0];
			//if(colt < parts[i].hit.t)
			{
				//parts[i].hit.t = colt;
				parts[i].willCollide = true;
				parts[i].nv[0] = -parts[i].v[0];
				parts[i].nv[1] = parts[i].v[1];
				parts[i].nv[2] = parts[i].v[2];
			}
			//parts[i].v[0] *= -1;
		}
		
		else if(parts[i].p[1]+parts[i].r+parts[i].v[1]*dt>SHEIGHT/2.0  || parts[i].p[1]-parts[i].r+parts[i].v[1]*dt<-SHEIGHT/2.0)
		{
			//colt = (SHEIGHT/2.0-fabs(parts[i].p[1])-parts[i].r)/parts[i].v[1];
			//if(colt < parts[i].hit.t)
			{
				//parts[i].hit.t = colt;
				parts[i].willCollide = true;
				parts[i].nv[0] = parts[i].v[0];
				parts[i].nv[1] = -parts[i].v[1];
				parts[i].nv[2] = parts[i].v[2];
			}
			//parts[i].v[1] *= -1;
		}
            }
	}
}

__global__
void advanceParticles(Particle *parts, int n, float dt, int gpuid, int BLOCK_SIZE, int els_for_each)
{


        int thid=blockIdx.x*BLOCK_SIZE+threadIdx.x;


	int i=thid+els_for_each*gpuid;
	{
		if(parts[i].willCollide == true)
		{
			parts[i].p[0] += parts[i].nv[0]*(dt);
			parts[i].p[1] += parts[i].nv[1]*(dt);
			parts[i].v[0] = parts[i].nv[0];
			parts[i].v[1] = parts[i].nv[1];
			parts[i].v[2] = parts[i].nv[2];
			parts[i].willCollide = false;
		}
		else
		{
			parts[i].p[0] += parts[i].v[0]*dt; 
			parts[i].p[1] += parts[i].v[1]*dt;
			//parts[i].p[2] += parts[i].v[2]*dt;
		}
	}
}     
	
  

// main routine that executes on the host
// >>>>>>>>> change main() -> cuda_main()
void* cuda_main(void* t) {
	long int gpuid = (long int)t;
	Particle *parts_h, *parts_d; // Pointer to host & device arrays
	const int N = els_for_each; // Number of elements in arrays
	size_t size = pNum * sizeof(Particle);
	int i;
	int deviceCount = 0;
	hipDeviceProp_t prop;
	int cur_dev;
        


	assert(hipSetDevice(gpuid)==hipSuccess);
	hipGetDevice(&cur_dev);
	printf("Dev: %d\n", cur_dev);


/*	hipGetDeviceCount(&deviceCount);

	for(i = 0; i < deviceCount; i++){
		hipGetDeviceProperties(&prop, i);
	}
*/
//	printf("%s.%d: The number of cuda devices is %d\n", __FUNCTION__, __LINE__, deviceCount);


	assert(hipMalloc((void **) &parts_d, size)==hipSuccess); // Allocate array on device
	printf("after hipMalloc: parts_d = %p\n", parts_d);
	// Initialize host array and copy it to CUDA device
	//for (int i = 0; i < N; i++)
	//	a_h[i] = (float) i;
	assert(hipMemcpy(parts_d, parts, size, hipMemcpyHostToDevice)==hipSuccess);
	// Do calculation on device:
	int block_size = 16;
	int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);

	for(int i=0; i<nIters; i++)
    	{
        	checkCollisionWithParticles<<<n_blocks,block_size>>>(parts_d, pNum, DT, gpuid, block_size, els_for_each);
        	checkCollisionWithBox<<<n_blocks,block_size>>>(parts_d, pNum, DT, gpuid, block_size, els_for_each);
        	advanceParticles<<<n_blocks,block_size>>>(parts_d, pNum, DT, gpuid, block_size, els_for_each);
		assert(hipMemcpy(parts+gpuid*els_for_each, parts_d+gpuid*els_for_each, els_for_each*sizeof(Particle), hipMemcpyDeviceToHost)==hipSuccess);
		printf("thid %d reached barier\n", gpuid);
                pthread_barrier_wait(&barr);
		printf("thid %d leaving barrier\n", gpuid);
                assert(hipMemcpy(parts_d, parts, size, hipMemcpyHostToDevice)==hipSuccess);


	}
	//hipConfigureCall(n_blocks, block_size, 0, 0);
	//hipSetupArgument(&a_d, sizeof(float*), 0);
	//hipSetupArgument(&N, sizeof(int), 0);
	//hipLaunchByPtr("square_array");	

	// Retrieve result from device and store it in host array
	//hipMemcpy(a_h, a_d, sizeof(float) * N, hipMemcpyDeviceToHost);
	
	// Cleanup
	//free(a_h);
	hipFree(parts_d);






	return 0;
}

void initParticles(Particle *parts, int n)
{
	for(int i=0; i<n; i++)
	{
		parts[i].p[0] = 300*(rand()/(float)RAND_MAX)-150;
		parts[i].p[1] = 300*(rand()/(float)RAND_MAX)-150;
		parts[i].p[2] = 0;// rand()/(float)RAND_MAX;
		
		parts[i].v[0] = rand()/(float)RAND_MAX - .5;
		parts[i].v[1] = rand()/(float)RAND_MAX - .5;
		parts[i].v[2] = 0;//rand()/(float)RAND_MAX - .5;
		
		parts[i].r = 2;
		parts[i].willCollide = false;
		parts[i].m = 100;
		//parts[i].hit.t = 100000;
	}

}


int main(int argv, char** argc){
	// I wonder, if this cannot be done by Python, since this is administration
	// and python integrates with c, so I think it might be wiser
	// to use python for that. but maybe later, when the thinks will clarify
	// create GPU assembly
	// 0. init if not initialized
	// (not here outthere) the device - create in buStore the representations
	// of the physical devices
	// 1. specify how many GPU you need
	// 2. create as many vgpu (in terms of structures) as required
	// 3. wire vgpu to gpus
	// 4. gpu assembly
	// 5. some process needs to clean up after - but we do not worry about that

	printf("\n*******************************\n");
	printf("CUDA APP START\n");
	printf("*******************************\n");
	
        hipError_t cuerr;
	double h_time=0;
	assert(argv>2);
	nIters = atoi(argc[1]);
	pNum = atoi(argc[2]);
	parts = (Particle *) malloc(pNum*sizeof(Particle));
	printf("Initializing particle system with pNum: %d\n", pNum);
        initParticles(parts, pNum);

	deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
        printf("NO OF DEVICES: %d\n", deviceCount);

	pthread_t* thid_array=(pthread_t*)malloc(deviceCount*sizeof(pthread_t));
	
	
	// now call the cuda main, so our GA enabler library can sort out
	// which cuda call goes where
	
	els_for_each = pNum/deviceCount;
	pthread_barrier_init(&barr, NULL, deviceCount);

	for(int i =0; i<deviceCount; i++)
	   pthread_create(&thid_array[i], NULL, cuda_main, (void*)i);
	
	//cuda_main();
	for(int i =0; i<deviceCount; i++)
	   pthread_join(thid_array[i], NULL);
	// Print results



	return 0;
}
