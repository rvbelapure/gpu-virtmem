#include "hip/hip_runtime.h"
/**
 * @file add1.cu
 * @brief this example is for testing cudaMemcpyFrom/ToSymbol
 *
 * @date Apr 27, 2011
 * @author Magda Slawinska, magg __at_ gatech __dot_ edu
 */


#include <stdio.h>
#include <hip/hip_runtime.h>

#define MAX 14

__device__ char name_device[MAX];
__device__ int tab_d[MAX];
__constant__ __device__ char hw[] = "Hello World!\n";

__global__ void helloWorldOnDevice(void) {
	int idx = blockIdx.x;
	name_device[idx] = hw[idx];
	tab_d[idx] *= tab_d[idx];
}

__global__ void inc(void){
	int idx = blockIdx.x;
	tab_d[idx]++;
}

int main(void) {
	int tab_h[MAX];
	int tab_h1[MAX];
	int i;
	char name_host[MAX];


	for (i = 0; i < MAX; i++)
		tab_h[i] = i;

	// symbol as a pointer
	hipMemcpyToSymbol(HIP_SYMBOL(tab_d), tab_h, sizeof(int) * MAX, 0,
			hipMemcpyHostToDevice);

	helloWorldOnDevice <<< MAX, 1 >>> ();
	hipDeviceSynchronize();

	// -----------  symbol as a pointer to a variable
	hipMemcpyFromSymbol(name_host, HIP_SYMBOL(name_device), sizeof(char) * 13, 0,
			hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(tab_h1, HIP_SYMBOL(tab_d), sizeof(int) * MAX, 0,
			hipMemcpyDeviceToHost);

	printf("\n\nGot from GPU: %s\n", name_host);
	if (strcmp(name_host, "Hello World!\n") == 0)
		printf("Hello test: PASSED\n");
	else
		printf("Hello test: FAILED\n");

	for (i = 0; i < MAX; i++) {
		if (tab_h1[i] != (tab_h[i] * tab_h[i])) {
			printf("FAILED!\n");
			break;
		} else
			printf("tab_h1[%d] = %d\n", i, tab_h1[i]);
	}

	// ----------- now symbol as a name
	// symbol as a name
	hipMemcpyToSymbol(HIP_SYMBOL("tab_d"), tab_h, sizeof(int) * MAX, 0,
				hipMemcpyHostToDevice);
	inc <<< MAX, 1 >>> ();
	hipDeviceSynchronize();
	hipMemcpyFromSymbol(tab_h1, HIP_SYMBOL("tab_d"), sizeof(int) * MAX, 0,
			hipMemcpyDeviceToHost);
	for (i = 0; i < MAX; i++) {
		if (tab_h1[i] != (tab_h[i] + 1)) {
			printf("FAILED!\n");
			break;
		} else
			printf("tab_h1[%d] = %d\n", i, tab_h1[i]);
	}

}

